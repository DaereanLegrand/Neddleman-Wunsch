#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <vector>
#include <string>
#include <algorithm>

// Scoring constants
const int MATCH = 2;
const int MISMATCH = -1;
const int GAP = -2;

// CUDA kernel for filling the scoring matrix
__global__ void fillMatrix(int* matrix, char* seq1, char* seq2, int len1, int len2, int pitch) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (i <= len1 && j <= len2) {
        int idx = i * pitch + j;
        int match = (seq1[i-1] == seq2[j-1]) ? MATCH : MISMATCH;
        int score = max(max(
            matrix[(i-1) * pitch + j] + GAP,
            matrix[i * pitch + (j-1)] + GAP),
            matrix[(i-1) * pitch + (j-1)] + match
        );
        matrix[idx] = score;
    }
}

// Function to perform Needleman-Wunsch alignment
std::vector<std::pair<std::string, std::string>> needlemanWunsch(const std::string& seq1, const std::string& seq2) {
    int len1 = seq1.length();
    int len2 = seq2.length();

    // Allocate and initialize device memory
    thrust::device_vector<char> d_seq1(seq1.begin(), seq1.end());
    thrust::device_vector<char> d_seq2(seq2.begin(), seq2.end());

    int pitch;
    int* d_matrix;
    hipMallocPitch(&d_matrix, (size_t*)&pitch, (len2 + 1) * sizeof(int), len1 + 1);
    pitch /= sizeof(int);

    // Initialize first row and column
    thrust::device_vector<int> d_init(len2 + 1);
    for (int i = 0; i <= len2; ++i) {
        d_init[i] = i * GAP;
    }
    hipMemcpy2D(d_matrix, pitch * sizeof(int), thrust::raw_pointer_cast(d_init.data()),
                 (len2 + 1) * sizeof(int), (len2 + 1) * sizeof(int), 1, hipMemcpyDeviceToDevice);

    for (int i = 1; i <= len1; ++i) {
        hipMemcpy(&d_matrix[i * pitch], &i, sizeof(int), hipMemcpyHostToDevice);
    }

    // Launch kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((len2 + blockSize.x - 1) / blockSize.x, (len1 + blockSize.y - 1) / blockSize.y);
    fillMatrix<<<gridSize, blockSize>>>(d_matrix, thrust::raw_pointer_cast(d_seq1.data()),
                                        thrust::raw_pointer_cast(d_seq2.data()),
                                        len1, len2, pitch);

    // Copy results back to host
    std::vector<int> h_matrix((len1 + 1) * (len2 + 1));
    hipMemcpy2D(h_matrix.data(), (len2 + 1) * sizeof(int), d_matrix,
                 pitch * sizeof(int), (len2 + 1) * sizeof(int), len1 + 1, hipMemcpyDeviceToHost);

    // Traceback to find alignments
    std::vector<std::pair<std::string, std::string>> alignments;
    std::function<void(int, int, std::string, std::string)> traceback = [&](int i, int j, std::string aligned1, std::string aligned2) {
        if (i == 0 && j == 0) {
            alignments.push_back({aligned1, aligned2});
            return;
        }

        int score = h_matrix[i * (len2 + 1) + j];
        int diag = (i > 0 && j > 0) ? h_matrix[(i-1) * (len2 + 1) + (j-1)] : INT_MIN;
        int up = (i > 0) ? h_matrix[(i-1) * (len2 + 1) + j] : INT_MIN;
        int left = (j > 0) ? h_matrix[i * (len2 + 1) + (j-1)] : INT_MIN;

        if (i > 0 && j > 0 && score == diag + ((seq1[i-1] == seq2[j-1]) ? MATCH : MISMATCH)) {
            traceback(i-1, j-1, seq1[i-1] + aligned1, seq2[j-1] + aligned2);
        }
        if (i > 0 && score == up + GAP) {
            traceback(i-1, j, seq1[i-1] + aligned1, '-' + aligned2);
        }
        if (j > 0 && score == left + GAP) {
            traceback(i, j-1, '-' + aligned1, seq2[j-1] + aligned2);
        }
    };

    traceback(len1, len2, "", "");

    // Clean up
    hipFree(d_matrix);

    return alignments;
}

// Function to return only one alignment
std::pair<std::string, std::string> needlemanWunschSingle(const std::string& seq1, const std::string& seq2) {
    auto alignments = needlemanWunsch(seq1, seq2);
    return alignments[0];
}

int main() {
    std::string bacteria = "tcaagcgtta gagaagtcat tatgtgataa aaaaattcaa cttggtatca acttaactaa gggtcttggt gctggtgctt tgcctgatgt tggtaaaggt gcagcagaag aatcaattga";
    std::string sars_cov = "attaaaggtt tataccttcc caggtaacaa accaaccaac tttcgatctc ttgtagatct gttctctaaa cgaactttaa aatctgtgtg gctgtcactc ggctgcatgc";
    std::string influenza = "atggaagcaa tatcactgat gactatacta ctggtggtaa caacaagtaa tgcagacaaa atctgcatcg gtcaccaatc aacaaattcc acggaaactg tagacacgct";

    // Perform alignments
    auto alignments1 = needlemanWunsch(bacteria, sars_cov);
    auto alignments2 = needlemanWunsch(sars_cov, influenza);
    auto alignments3 = needlemanWunsch(bacteria, influenza);

    // Print results
    std::cout << "Bacteria vs SARS-CoV alignments: " << alignments1.size() << std::endl;
    std::cout << "SARS-CoV vs Influenza alignments: " << alignments2.size() << std::endl;
    std::cout << "Bacteria vs Influenza alignments: " << alignments3.size() << std::endl;

    // Example of using single alignment function
    auto single_alignment = needlemanWunschSingle(bacteria, sars_cov);
    std::cout << "Single alignment example:" << std::endl;
    std::cout << single_alignment.first << std::endl;
    std::cout << single_alignment.second << std::endl;

    return 0;
}
